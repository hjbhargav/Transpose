#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

const int N= 1024; // matrix size is NxN
const int K= 32; // tile size is KxK

void fill_matrix(float *mat)
{
for(int j=0; j < N * N; j++)
mat[j] = (float) j;
}

__global__ void transpose_serial(float in[], float out[])
{
for(int j=0; j < N; j++)
for(int i=0; i < N; i++)
out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

__global__ void transpose_parallel_per_row(float in[], float out[])
{
int i = threadIdx.x;
for(int j=0; j < N; j++)
out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

__global__ void transpose_parallel_per_element(float in[], float out[])
{
int i = blockIdx.x * K + threadIdx.x;
int j = blockIdx.y * K + threadIdx.y;
out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

__global__ void transpose_parallel_per_element_tiled16(float in[], float out[])
{
int in_corner_i = blockIdx.x * 16, in_corner_j = blockIdx.y * 16;
int out_corner_i = blockIdx.y * 16, out_corner_j = blockIdx.x * 16;
int x = threadIdx.x, y = threadIdx.y;
__shared__ float tile[16][16];
// coalesced read from global mem, TRANSPOSED write into shared mem:
tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
__syncthreads();
// read from shared mem, coalesced write to global mem:
out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

int main(int argc, char **argv)
{
int numbytes = N * N * sizeof(float);
float *in = (float *) malloc(numbytes);
float *out = (float *) malloc(numbytes);
fill_matrix(in);
float *d_in, *d_out;
hipMalloc(&d_in, numbytes);
hipMalloc(&d_out, numbytes);
hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);
GpuTimer timer;

timer.Start();
transpose_serial<<<1,1>>>(d_in, d_out);
timer.Stop();
printf("Transpose_serial: %g ms.\n",timer.Elapsed());

timer.Start();
transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
timer.Stop();
printf("Transpose_per_row: %g ms.\n",timer.Elapsed());

dim3 blocks(N/K,N/K); // blocks per grid
dim3 threads(K,K); // threads per block
timer.Start();
transpose_parallel_per_element<<<blocks,threads>>>(d_in, d_out);
timer.Stop();
printf("Transpose_per_element: %g ms.\n",timer.Elapsed());

dim3 blocks16x16(N/16,N/16); // blocks per grid
dim3 threads16x16(16,16); // threads per block
timer.Start();
transpose_parallel_per_element_tiled16<<<blocks16x16,threads16x16>>>(d_in, d_out);
timer.Stop();
printf("Transpose_tiled: %g ms.\n",timer.Elapsed());
return 0;
}
